#include "hip/hip_runtime.h"
#include "config.h"
#include "util.h"
#include <climits>
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <ctime>
#include <hip/hip_runtime.h>
#include <string>

#ifndef BLOCK_DIM
#define BLOCK_DIM 16
#endif

void floydWarshall(int *matrix, const int n);
__global__ void floyd_warshall_block_kernel_phase1(int n, int k, int *graph);
__global__ void floyd_warshall_block_kernel_phase2(int n, int k, int *graph);
__global__ void floyd_warshall_block_kernel_phase3(int n, int k, int *graph);

int main(int argc, char *argv[]) {
    int n, density, threadsPerBlock;

    threadsPerBlock = BLOCK_DIM * BLOCK_DIM;
    if (argc <= 2) {
        n = DEFAULT;
        density = 100;
    } else {
        n = atoi(argv[1]);
        density = atoi(argv[2]);
    }

    int *matrix = (int *)malloc(n * n * sizeof(int));
    populateMatrix(matrix, n, density);

    if (PRINTABLE) {
        printf("*** Adjacency matrix:\n");
        showDistances(matrix, n);
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    floydWarshall(matrix, n);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float accum = 0;
    hipEventElapsedTime(&accum, start, stop);

    if (PRINTABLE) {
        printf("*** The solution is:\n");
        showDistances(matrix, n);
    }

    printf("[GPGPU] Total elapsed time %.2f ms\n", accum);
    free(matrix);

    // calculate theoretical occupancy
    int maxActiveBlocksPerSM, device;
    float occupancy;
    hipDeviceProp_t props;
    hipGetDevice(&device);
    hipGetDeviceProperties(&props, device);

    hipOccupancyMaxActiveBlocksPerMultiprocessor(&maxActiveBlocksPerSM,
                                                  floyd_warshall_block_kernel_phase1, threadsPerBlock,
                                                  0);
    occupancy = (maxActiveBlocksPerSM * threadsPerBlock / props.warpSize) /
                (float)(props.maxThreadsPerMultiProcessor /
                        props.warpSize);
    printf("Phase: 1\n");
    printf("maxActiveBlocksPerSM: %d, warpSize: %d, maxThreadsPerMultiProcessor: %d\n", maxActiveBlocksPerSM, props.warpSize, props.maxThreadsPerMultiProcessor);
    printf("Launched blocks of size: %d, Theoretical occupancy: %f\n", threadsPerBlock, occupancy);

    hipOccupancyMaxActiveBlocksPerMultiprocessor(&maxActiveBlocksPerSM,
                                                  floyd_warshall_block_kernel_phase1, threadsPerBlock,
                                                  0);
    occupancy = (maxActiveBlocksPerSM * threadsPerBlock / props.warpSize) /
                (float)(props.maxThreadsPerMultiProcessor /
                        props.warpSize);
    printf("Phase: 2\n");
    printf("maxActiveBlocksPerSM: %d, warpSize: %d, maxThreadsPerMultiProcessor: %d\n", maxActiveBlocksPerSM, props.warpSize, props.maxThreadsPerMultiProcessor);
    printf("Launched blocks of size: %d, Theoretical occupancy: %f\n", threadsPerBlock, occupancy);

    hipOccupancyMaxActiveBlocksPerMultiprocessor(&maxActiveBlocksPerSM,
                                                  floyd_warshall_block_kernel_phase1, threadsPerBlock,
                                                  0);
    occupancy = (maxActiveBlocksPerSM * threadsPerBlock / props.warpSize) /
                (float)(props.maxThreadsPerMultiProcessor /
                        props.warpSize);
    printf("Phase: 3\n");
    printf("maxActiveBlocksPerSM: %d, warpSize: %d, maxThreadsPerMultiProcessor: %d\n", maxActiveBlocksPerSM, props.warpSize, props.maxThreadsPerMultiProcessor);
    printf("Launched blocks of size: %d, Theoretical occupancy: %f\n", threadsPerBlock, occupancy);

    FILE *pFile;
    pFile = fopen("./result/cuda_block_result.txt", "a");
    if (NULL == pFile) {
        puts("open failure");
        exit(1);
    } else {
        fprintf(pFile, "%d\t%d\t%d\t%.2f\n", threadsPerBlock, n, density, accum);
    }
    fclose(pFile);

    return 0;
}

void floydWarshall(int *matrix, const int n) {
    int *deviceMatrix;
    int size = n * n * sizeof(int);
    hipMalloc(&deviceMatrix, size);
    hipMemcpy(deviceMatrix, matrix, size, hipMemcpyHostToDevice);

    const int blocks = (n + BLOCK_DIM - 1) / BLOCK_DIM;
    dim3 dimBlock(BLOCK_DIM, BLOCK_DIM, 1);
    dim3 dimGrid(blocks, blocks, 1);

    for (int k = 0; k < blocks; k++) {
        floyd_warshall_block_kernel_phase1<<<1, dimBlock>>>(n, k, deviceMatrix);
        floyd_warshall_block_kernel_phase2<<<blocks, dimBlock>>>(n, k, deviceMatrix);
        floyd_warshall_block_kernel_phase3<<<dimGrid, dimBlock>>>(n, k, deviceMatrix);
    }

    hipMemcpy(matrix, deviceMatrix, size, hipMemcpyDeviceToHost);
    hipFree(deviceMatrix);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(err), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }
}

__forceinline__
    __device__ void
    block_calc(int *C, int *A, int *B, int bi, int bj) {
    for (int k = 0; k < BLOCK_DIM; k++) {
        int sum = A[bi * BLOCK_DIM + k] + B[k * BLOCK_DIM + bj];
        if (C[bi * BLOCK_DIM + bj] > sum) {
            C[bi * BLOCK_DIM + bj] = sum;
        }
        __syncthreads();
    }
}

__global__ void floyd_warshall_block_kernel_phase1(int n, int k, int *graph) {
    const int bi = threadIdx.y;
    const int bj = threadIdx.x;
    const int row_idx = k * BLOCK_DIM + bi;
    const int col_idx = k * BLOCK_DIM + bj;

    __shared__ int C[BLOCK_DIM * BLOCK_DIM];

    // Transfer to temp shared arrays
    C[bi * BLOCK_DIM + bj] = graph[row_idx * n + col_idx];

    __syncthreads();

    block_calc(C, C, C, bi, bj);
    // Transfer back to graph
    graph[row_idx * n + col_idx] = C[bi * BLOCK_DIM + bj];
}

__global__ void floyd_warshall_block_kernel_phase2(int n, int k, int *graph) {
    // BlockDim is one dimensional (Straight along diagonal)
    // Blocks themselves are two dimensional
    const int i = blockIdx.x;
    const int bi = threadIdx.y;
    const int bj = threadIdx.x;

    if (i == k)
        return;

    __shared__ int A[BLOCK_DIM * BLOCK_DIM];
    __shared__ int B[BLOCK_DIM * BLOCK_DIM];
    __shared__ int C[BLOCK_DIM * BLOCK_DIM];

    C[bi * BLOCK_DIM + bj] = graph[i * BLOCK_DIM * n + k * BLOCK_DIM + bi * n + bj];
    B[bi * BLOCK_DIM + bj] = graph[k * BLOCK_DIM * n + k * BLOCK_DIM + bi * n + bj];

    __syncthreads();

    block_calc(C, C, B, bi, bj);
    graph[i * BLOCK_DIM * n + k * BLOCK_DIM + bi * n + bj] = C[bi * BLOCK_DIM + bj];

    __syncthreads();

    C[bi * BLOCK_DIM + bj] = graph[k * BLOCK_DIM * n + i * BLOCK_DIM + bi * n + bj];
    A[bi * BLOCK_DIM + bj] = graph[k * BLOCK_DIM * n + k * BLOCK_DIM + bi * n + bj];

    __syncthreads();

    block_calc(C, A, C, bi, bj);
    // Block C is the only one that could be changed
    graph[k * BLOCK_DIM * n + i * BLOCK_DIM + bi * n + bj] = C[bi * BLOCK_DIM + bj];
}

__global__ void floyd_warshall_block_kernel_phase3(int n, int k, int *graph) {
    // BlockDim is one dimensional (Straight along diagonal)
    // Blocks themselves are two dimensional
    const int j = blockIdx.x;
    const int i = blockIdx.y;
    const int bi = threadIdx.y;
    const int bj = threadIdx.x;

    if (i == k && j == k)
        return;

    __shared__ int A[BLOCK_DIM * BLOCK_DIM];
    __shared__ int B[BLOCK_DIM * BLOCK_DIM];
    __shared__ int C[BLOCK_DIM * BLOCK_DIM];

    C[bi * BLOCK_DIM + bj] = graph[i * BLOCK_DIM * n + j * BLOCK_DIM + bi * n + bj];
    A[bi * BLOCK_DIM + bj] = graph[i * BLOCK_DIM * n + k * BLOCK_DIM + bi * n + bj];
    B[bi * BLOCK_DIM + bj] = graph[k * BLOCK_DIM * n + j * BLOCK_DIM + bi * n + bj];

    __syncthreads();

    block_calc(C, A, B, bi, bj);
    graph[i * BLOCK_DIM * n + j * BLOCK_DIM + bi * n + bj] = C[bi * BLOCK_DIM + bj];
}
