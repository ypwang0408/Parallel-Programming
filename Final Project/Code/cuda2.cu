#include "hip/hip_runtime.h"
// nvcc cuda2.cu -o cuda2.out -gencode=arch=compute_75,code=compute_75 -O3
#include "config.h"
#include "util.h"
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <ctime>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <string>

#define BLOCK_SIZE 4

__global__ void wakeGPU(int reps);
__global__ void floydWarshallKernel(int k, int *matrix, int n);

void floydWarshall(int *matrix, int n, int threadsPerBlock);

int main(int argc, char *argv[]) {
    int n, density, threadsPerBlock;

    if (argc <= 3) {
        n = DEFAULT;
        density = 100;
        threadsPerBlock = BLOCK_SIZE;
    } else {
        n = atoi(argv[1]);
        density = atoi(argv[2]);
        threadsPerBlock = atoi(argv[3]);
    }

    int *matrix = (int *)malloc(n * n * sizeof(int));

    populateMatrix(matrix, n, density);

    if (PRINTABLE) {
        printf("*** Adjacency matrix:\n");
        showDistances(matrix, n);
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    floydWarshall(matrix, n, threadsPerBlock);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float accum;
    hipEventElapsedTime(&accum, start, stop);

    if (PRINTABLE) {
        printf("*** The solution is:\n");
        showDistances(matrix, n);
    }

    printf("[GPGPU] Total elapsed time %.2f ms\n", accum);

    // calculate theoretical occupancy
    int maxActiveBlocksPerSM;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&maxActiveBlocksPerSM,
                                                  floydWarshallKernel,
                                                  threadsPerBlock,
                                                  0);

    int device;
    hipDeviceProp_t props;
    hipGetDevice(&device);
    hipGetDeviceProperties(&props, device);

    float occupancy = (maxActiveBlocksPerSM * threadsPerBlock / props.warpSize) /
                      (float)(props.maxThreadsPerMultiProcessor /
                              props.warpSize);

    printf("Launched blocks of size %d. Theoretical occupancy: %f\n",
           threadsPerBlock, occupancy);

    free(matrix);

    FILE *pFile;
    pFile = fopen("./result/cuda2_result.txt", "a");
    if (NULL == pFile) {
        puts("open failure");
        exit(1);
    } else {
        fprintf(pFile, "%d\t%d\t%d\t%.2f\n", threadsPerBlock, n, density, accum);
    }
    fclose(pFile);

    return 0;
}

void floydWarshall(int *matrix, const int n, int threadsPerBlock) {
    int *deviceMatrix;
    int size = n * n * sizeof(int);

    hipMalloc((int **)&deviceMatrix, size);
    hipMemcpy(deviceMatrix, matrix, size, hipMemcpyHostToDevice);

    dim3 dimGrid((n + threadsPerBlock - 1) / threadsPerBlock, n);

    hipFuncSetCacheConfig(reinterpret_cast<const void*>(floydWarshallKernel), hipFuncCachePreferL1);
    for (int k = 0; k < n; k++) {
        floydWarshallKernel<<<dimGrid, threadsPerBlock>>>(k, deviceMatrix, n);
    }
    hipDeviceSynchronize();

    hipMemcpy(matrix, deviceMatrix, size, hipMemcpyDeviceToHost);

    hipFree(deviceMatrix);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(err), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }
}

__global__ void floydWarshallKernel(int k, int *matrix, int n) {
    int i = blockDim.y * blockIdx.y;
    int j = blockDim.x * blockIdx.x + threadIdx.x;

    if (j < n) {
        int newPath = matrix[k * n + j] + matrix[i * n + k];
        int oldPath = matrix[i * n + j];
        if (oldPath > newPath) {
            matrix[i * n + j] = newPath;
        }
    }
}